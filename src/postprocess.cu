#include <hip/hip_runtime.h>
#include <algorithm>


static __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy);

static __global__ void decode_kernel(
    float *predict, int NUM_BOX_ELEMENT, int num_bboxes, int num_classes,
    int ckpt, float confidence_threshold, float *invert_affine_matrix,
    float *parray, int max_objects);

static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom,
    float bleft, float btop, float bright, float bbottom);

static __global__ void nms_kernel(float *bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT);

void decode_kernel_invoker(
    float *predict, int NUM_BOX_ELEMENT, int num_bboxes, int num_classes,
    int ckpt, float confidence_threshold, float *invert_affine_matrix,
    float *parray, int max_objects, hipStream_t stream)
{
    // int block = 256;
    // int grid =  ceil(num_bboxes / (float)block);
    dim3 dimblock(256, 1, 1);
    dim3 dimgird((num_bboxes + dimblock.x - 1) / dimblock.x, 1, 1);

    decode_kernel<<<dimgird, dimblock, 0, stream>>>(
        predict, NUM_BOX_ELEMENT, num_bboxes, num_classes,
        ckpt, confidence_threshold, invert_affine_matrix,
        parray, max_objects);
}

void nms_kernel_invoker(float *parray, float nms_threshold, int max_objects, hipStream_t stream, int NUM_BOX_ELEMENT)
{
    int block = std::min(256, max_objects);
    int grid = std::ceil(1.0 * max_objects / block);
    // int grid = 32;
    nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold, NUM_BOX_ELEMENT);
}

static __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy)
{
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel(
    float *predict, int NUM_BOX_ELEMENT, int num_bboxes, int num_classes,
    int ckpt, float confidence_threshold, float *invert_affine_matrix,
    float *parray, int max_objects)
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes)
        return;

    float *pitem = predict + (5 + num_classes + ckpt * 2) * position;
    // v7: cx, cy, w, h, conf, x1, y1, conf1, x2, y2, conf2, x3, y3, conf3, x4, y4, conf4, conf_c1...conf_c36
    // v5: cx, cy, w, h, conf, x1, y1, x2, y2, x3, y3, x4, y4, conf_c1...conf_c36
    float objectness = pitem[4];
    if (objectness < confidence_threshold)
        return;

    float *class_confidence = pitem + 5 + ckpt * 2;
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence)
    {
        if (*class_confidence > confidence)
        {
            confidence = *class_confidence;
            label = i;
        }
    }

    confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= max_objects)
        return;

    float cx = pitem[0];
    float cy = pitem[1];
    float width = pitem[2];
    float height = pitem[3];

    // 4个关键点
    float *landmarks = pitem + 5;
    float x1 = landmarks[0];
    float y1 = landmarks[1];
    float x2 = landmarks[2];
    float y2 = landmarks[3];
    float x3 = landmarks[4];
    float y3 = landmarks[5];
    float x4 = landmarks[6];
    float y4 = landmarks[7];

    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    affine_project(invert_affine_matrix, x1, y1, &x1, &y1);
    affine_project(invert_affine_matrix, x2, y2, &x2, &y2);
    affine_project(invert_affine_matrix, x3, y3, &x3, &y3);
    affine_project(invert_affine_matrix, x4, y4, &x4, &y4);

    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore

    *pout_item++ = x1;
    *pout_item++ = y1;

    *pout_item++ = x2;
    *pout_item++ = y2;

    *pout_item++ = x3;
    *pout_item++ = y3;

    *pout_item++ = x4;
    *pout_item++ = y4;
}

static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom,
    float bleft, float btop, float bright, float bbottom)
{

    float cleft = std::max(aleft, bleft);
    float ctop = std::max(atop, btop);
    float cright = std::min(aright, bright);
    float cbottom = std::min(abottom, bbottom);

    float c_area = std::max(cright - cleft, 0.0f) * std::max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;

    float a_area = std::max(0.0f, aright - aleft) * std::max(0.0f, abottom - atop);
    float b_area = std::max(0.0f, bright - bleft) * std::max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void nms_kernel(float *bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT)
{

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = std::min((int)*bboxes, max_objects);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i)
    {
        float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5])
            continue;

        if (pitem[4] >= pcurrent[4])
        {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0], pitem[1], pitem[2], pitem[3]);

            if (iou > threshold)
            {
                pcurrent[6] = 0; // 1=keep, 0=ignore
                return;
            }
        }
    }
}